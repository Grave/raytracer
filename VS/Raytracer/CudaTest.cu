#include "hip/hip_runtime.h"
#include "CudaTest.h"

#include <iostream> 
#include <hip/hip_runtime.h> 
 
#pragma comment(lib, "cudart") 
 
using std::cerr; 
using std::cout; 
using std::endl; 
using std::exception; 
 
const int CudaTest::MaxSize = 96; 
 
// CUDA kernel: cubes each array value 
__global__ void cubeKernel(float* result, float* data) 
{ 
    int idx = threadIdx.x; 
    float f = data[idx]; 
    result[idx] = f * f * f; 
} 
 
// Initializes data on the host 
void CudaTest::InitializeData(vector<float>& data) 
{ 
    for (int i = 0; i < MaxSize; ++i) 
    { 
        data[i] = static_cast<float>(i); 
    } 
} 
 
// Executes CUDA kernel 
void CudaTest::RunCubeKernel(vector<float>& data, vector<float>& result) 
{ 
    const size_t size = MaxSize * sizeof(float); 
 
    // TODO: test for error 
    float* d; 
    float* r; 
    hipError_t hr; 
 
    hr = hipMalloc(reinterpret_cast<void**>(&d), size);            // Could return 46 if device is unavailable. 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    hr = hipMalloc(reinterpret_cast<void**>(&r), size); 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    // Copy data to the device 
    hipMemcpy(d, &data[0], size, hipMemcpyHostToDevice); 
 
    // Launch kernel: 1 block, 96 threads 
    // Important: Do not exceed number of threads returned by the device query, 1024 on my computer. 
    cubeKernel<<<1, MaxSize>>>(r, d); 
 
    // Copy back to the host 
    hipMemcpy(&result[0], r, size, hipMemcpyDeviceToHost); 
 
    // Free device memory 
    hipFree(d); 
    hipFree(r); 
}